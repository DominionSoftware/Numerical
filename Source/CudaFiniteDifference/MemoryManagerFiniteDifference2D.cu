#include "MemoryManagerFiniteDifference2D.cuh"

#include <stdexcept>


using namespace numerical;

MemoryManagerFiniteDifference2D::MemoryManagerFiniteDifference2D(size_t sz)
{
	inputData_.allocate(sz);
	outputData_.allocate(sz);

}

void MemoryManagerFiniteDifference2D::copyHostInputDataToDevice(float* data, size_t sz)
{
	hipError_t error = hipMemcpy(getInputData(), data, sz, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		throw std::runtime_error(hipGetErrorString(error));
	}
}

void MemoryManagerFiniteDifference2D::copyDeviceOutputDataToHost(float* data, size_t sz)
{
	hipError_t error = hipMemcpy(data,getOutputData(), sz, hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		throw std::runtime_error(hipGetErrorString(error));
	}
}
